#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


//thread 1D
__global__ void testThread1(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = b[i] - a[i];
}

//thread 2D
__global__ void testThread2(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + threadIdx.y*blockDim.x;
    c[i] = b[i] - a[i];
}

//thread 3D
__global__ void testThread3(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    c[i] = b[i] - a[i];
}

//block 1D
__global__ void testBlock1(int *c, const int *a, const int *b)
{
    int i = blockIdx.x;
    c[i] = b[i] - a[i];
}

//block 2D
__global__ void testBlock2(int *c, const int *a, const int *b)
{
    int i = blockIdx.x + blockIdx.y*gridDim.x;
    c[i] = b[i] - a[i];
}

//block 3D
__global__ void testBlock3(int *c, const int *a, const int *b)
{
    int i = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    c[i] = b[i] - a[i];
}

//block-thread 1D-1D
__global__ void testBlockThread1(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    c[i] = b[i] - a[i];
}

//block-thread 1D-2D
__global__ void testBlockThread2(int *c, const int *a, const int *b)
{
    int threadId_2D = threadIdx.x + threadIdx.y*blockDim.x;
    int i = threadId_2D+ (blockDim.x*blockDim.y)*blockIdx.x;
    c[i] = b[i] - a[i];
}

//block-thread 1D-3D
__global__ void testBlockThread3(int *c, const int *a, const int *b)
{
    int threadId_3D = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    int i = threadId_3D + (blockDim.x*blockDim.y*blockDim.z)*blockIdx.x;
    c[i] = b[i] - a[i];
}
//uint3 s1; s1.x = 10; s1.y = 10; s1.z = 1;
//uint3 s2; s2.x = size / 100; s2.y = 1; s2.z = 1;
//testBlockThread4 << <s1, s2 >> >(dev_c, dev_a, dev_b);
//block-thread 2D-1D
__global__ void testBlockThread4(int *c, const int *a, const int *b)
{
    int blockId_2D = blockIdx.x + blockIdx.y*gridDim.x;
    int i = threadIdx.x + blockDim.x*blockId_2D;
    c[i] = b[i] - a[i];
}

//block-thread 3D-1D
__global__ void testBlockThread5(int *c, const int *a, const int *b)
{
    int blockId_3D = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int i = threadIdx.x + blockDim.x*blockId_3D;
    c[i] = b[i] - a[i];
}

//block-thread 2D-2D
__global__ void testBlockThread6(int *c, const int *a, const int *b)
{
    int threadId_2D = threadIdx.x + threadIdx.y*blockDim.x;
    int blockId_2D = blockIdx.x + blockIdx.y*gridDim.x;
    int i = threadId_2D + (blockDim.x*blockDim.y)*blockId_2D;
    c[i] = b[i] - a[i];
}

//block-thread 2D-3D
__global__ void testBlockThread7(int *c, const int *a, const int *b)
{
    int threadId_3D = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    int blockId_2D = blockIdx.x + blockIdx.y*gridDim.x;
    int i = threadId_3D + (blockDim.x*blockDim.y*blockDim.z)*blockId_2D;
    c[i] = b[i] - a[i];
}

//block-thread 3D-2D
__global__ void testBlockThread8(int *c, const int *a, const int *b)
{
    int threadId_2D = threadIdx.x + threadIdx.y*blockDim.x;
    int blockId_3D = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int i = threadId_2D + (blockDim.x*blockDim.y)*blockId_3D;
    c[i] = b[i] - a[i];
}

//block-thread 3D-3D
__global__ void testBlockThread9(int *c, const int *a, const int *b)
{
    int threadId_3D = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;
    int blockId_3D = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int i = threadId_3D + (blockDim.x*blockDim.y*blockDim.z)*blockId_3D;
    c[i] = b[i] - a[i];
}


// cuda 单元从大到小 grid->blocks->threads
//cuda 通过<<< >>>符号来分配索引线程的方式，我知道的一共有15种索引方式。
extern "C" void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    //testThread1<<<1, size>>>(dev_c, dev_a, dev_b);

    //uint3 s;
    //s.x = size/5;
    //s.y = 5;
    //s.z = 1;
    //testThread2 <<<1,s>>>(dev_c, dev_a, dev_b);

    //uint3 s;
    //s.x = size / 10;
    //s.y = 5;
    //s.z = 2;
    //testThread3<<<1, s >>>(dev_c, dev_a, dev_b);

    //testBlock1<<<size,1 >>>(dev_c, dev_a, dev_b);

    //uint3 s;
    //s.x = size / 5;
    //s.y = 5;
    //s.z = 1;
    //testBlock2<<<s, 1 >>>(dev_c, dev_a, dev_b);

    //uint3 s;
    //s.x = size / 10;
    //s.y = 5;
    //s.z = 2;
    //testBlock3<<<s, 1 >>>(dev_c, dev_a, dev_b);

    //testBlockThread1<<<size/10, 10>>>(dev_c, dev_a, dev_b);

    //uint3 s1;
    //s1.x = size / 100;
    //s1.y = 1;
    //s1.z = 1;
    //uint3 s2;
   // s2.x = 10;
   // s2.y = 10;
   // s2.z = 1;
    //testBlockThread2 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1;
    //s1.x = size / 100;
    //s1.y = 1;
    //s1.z = 1;
    //uint3 s2;
    //s2.x = 10;
    //s2.y = 5;
    //s2.z = 2;
    //testBlockThread3 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1;
    //s1.x = 10;
   // s1.y = 10;
   // s1.z = 1;
    //uint3 s2;
    // s2.x = size / 100;
    // s2.y = 1;
    // s2.z = 1;
    //testBlockThread4 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1;
    //s1.x = 10;
    //s1.y = 5;
    //s1.z = 2;
    //uint3 s2;
    // s2.x = size / 100;
    // s2.y = 1;
    // s2.z = 1;
    //testBlockThread5 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1;
    // s1.x = size / 100;
    //s1.y = 10;
    //s1.z = 1;
    //uint3 s2;
    //s2.x = 5;
    //s2.y = 2;
    //s2.z = 1;
    //testBlockThread6 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1;
    //s1.x = size / 100;
    //s1.y = 5;
    //s1.z = 1;
    //uint3 s2;
    //s2.x = 5;
    //s2.y = 2;
    //s2.z = 2;
    //testBlockThread7 << <s1, s2 >> >(dev_c, dev_a, dev_b);

    //uint3 s1;
    //s1.x = 5;
    //s1.y = 2;
    //s1.z = 2;
    //uint3 s2;
    //s2.x = size / 100;
    //s2.y = 5;
    //s2.z = 1;
    //testBlockThread8 <<<s1, s2 >>>(dev_c, dev_a, dev_b);

    uint3 s1; s1.x = 5; s1.y = 2; s1.z = 2;
    uint3 s2; s2.x = size / 200; s2.y = 5; s2.z = 2;
    testBlockThread9<<<s1, s2 >>>(dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, size*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipGetLastError();
}
