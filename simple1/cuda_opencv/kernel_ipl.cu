#ifndef _KERNEL_IPL_CU_
#define _KERNEL_IPL_CU_

#include<time.h>

#include "hip/hip_runtime.h"



#define BYTE unsigned char

__global__ void InverseImg_kernel(BYTE* pImgOut, BYTE* pImgIn, int nWidth, int nHeight, int nWidthStep)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nWidth && iy < nHeight)
    {
        pImgOut[iy * nWidthStep + ix] =
            255 - pImgIn[iy * nWidthStep + ix];
    }

}

extern "C"
double cudaInverseImg(BYTE* pImgOut, BYTE* pImgIn, int nWidth, int nHeight, int nWidthStep, int nChannels)
{
    // var for timing
    clock_t start, finish;
    double  duration = 0.0;

    // cpu 计时开始
    start = clock();

    // 准备空间
    BYTE* d_pImgOut;
    BYTE* d_pImgIn;
    hipMalloc((void**)&d_pImgOut, sizeof(BYTE) * nWidthStep * nHeight);
    hipMalloc((void**)&d_pImgIn, sizeof(BYTE) * nWidthStep * nHeight);

    //传入数据源
    hipMemcpy(d_pImgIn, pImgIn, sizeof(BYTE) * nWidthStep * nHeight, hipMemcpyHostToDevice);

    hipMemset(d_pImgOut, 0, sizeof(BYTE) * nWidthStep * nHeight);

    //ＧＰＵ处理
    dim3 ts(16, 16);
    dim3 bs((nWidth*nChannels + 15) / 16, (nHeight + 15) / 16);
    InverseImg_kernel<<< bs, ts >>>(d_pImgOut, d_pImgIn, nWidth*nChannels, nHeight, nWidthStep);


    //输出结果
    hipMemcpy(pImgOut, d_pImgOut, sizeof(BYTE) * nWidthStep * nHeight, hipMemcpyDeviceToHost);

    //释放空间
    hipFree(d_pImgOut);
    hipFree(d_pImgIn);

    //cpu 计时结束
    finish = clock();
    duration = (double)(finish - start) / CLOCKS_PER_SEC;

    return duration;
}

#endif
